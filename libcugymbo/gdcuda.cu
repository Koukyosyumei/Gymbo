#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "CudaVecDouble.h"

__global__ void kernel_vecDouble(int *in, int *out, const int n)
{
    int i = threadIdx.x;
    if (i < n) {
        out[i] = in[i] * 2;
    }
}

void vecDouble(int *hIn, int *hOut, const int n)
{
    int *dIn;
    int *dOut;
    hipHostMalloc((void**)&dIn, n * sizeof(int));
    hipHostMalloc((void**)&dOut, n * sizeof(int));
    hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

    kernel_vecDouble<<<1, n>>>(dIn, dOut, n);
    hipDeviceSynchronize();

    hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dIn);
    hipFree(dOut);
}
